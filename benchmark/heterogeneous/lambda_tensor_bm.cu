﻿#include <benchmark/benchmark.h>
#include <bm_config.hpp>
#include <matazure/tensor>

using namespace matazure;

#ifdef USE_CUDA

static void BM_linear_lambda_tensor_persist_gold(benchmark::State &st) {
	tensor<float, 1> tsf1(st.range(0));
	while (st.KeepRunning()) {
		tensor<float, 1> ts_re(tsf1.shape());
		for (int_t i = 0; i < ts_re.size(); ++i) {
			ts_re[i] = 2.0f * tsf1[i];
		}
	}

	auto bytes_size = static_cast<size_t>(tsf1.size()) * sizeof(decltype(tsf1[0]));
	st.SetBytesProcessed(st.iterations() * bytes_size);
}

static void BM_linear_lambda_tensor_persist(benchmark::State &st) {
	tensor<float, 1> tsf1(st.range(0));
	while (st.KeepRunning()) {
		auto tsf1_re = make_lambda(tsf1.shape(), [tsf1](int_t i) {
			return 2.0f * tsf1[i];
		}).persist();
	}

	auto bytes_size = static_cast<size_t>(tsf1.size()) * sizeof(decltype(tsf1[0]));
	st.SetBytesProcessed(st.iterations() * bytes_size);
}

BENCHMARK(BM_linear_lambda_tensor_persist_gold)->Range(1 << 10, 1 << (bm_config::max_host_memory_exponent() - 2))->UseRealTime();
BENCHMARK(BM_linear_lambda_tensor_persist)->Range(1 << 10, 1 << (bm_config::max_host_memory_exponent() - 2))->UseRealTime();

#endif
