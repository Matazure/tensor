﻿#include <benchmark/benchmark.h>
#include <matazure/tensor>
#include <matazure/cuda/puzzle/conv.hpp>

using namespace matazure;

static_tensor<float,meta::array< 3,  3>> host_mask;
__constant__ static_tensor<float,meta::array< 3,  3>> mask;
MATAZURE_PUZZEL_CONV_GLOBAL(conv_global, mask)
MATAZURE_PUZZEL_CONV_BLOCK(conv_block, mask)
MATAZURE_PUZZEL_CONV_BLOCK_WITH_CRACK(conv_block_crack, mask)

template <typename _ValueType>
void BM_cu_conv_global(benchmark::State& state) {
	pointi<2> ext;
	fill(ext, state.range(0));
	cu_tensor<_ValueType, 2> ts_src(ext);
	cu_tensor<_ValueType, 2> ts_re(ts_src.extent());
	fill(host_mask, 1.0f / host_mask.size());
	cuda::copy_symbol(host_mask, mask);

	while (state.KeepRunning()) {
		copy(cuda::puzzle::conv_global(clamp_zero(ts_src)), ts_re);
		cuda::barrier();
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size * 2);
}
BENCHMARK_TEMPLATE1(BM_cu_conv_global, float)->RangeMultiplier(2)->Range(128, 4096)->UseRealTime();

template <typename _ValueType>
void BM_cu_conv_block(benchmark::State& state) {
	pointi<2> ext;
	fill(ext, state.range(0));
	cu_tensor<_ValueType, 2> ts_src(ext);
	cu_tensor<_ValueType, 2> ts_re(ts_src.extent());
	fill(host_mask, 1.0f / host_mask.size());
	cuda::copy_symbol(host_mask, mask);

	while (state.KeepRunning()) {
		cuda::puzzle::conv_block<16,16>(clamp_zero(ts_src), ts_re);
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size * 2);
}
BENCHMARK_TEMPLATE1(BM_cu_conv_block, float)->RangeMultiplier(2)->Range(128, 4096)->UseRealTime();

template <typename _ValueType>
void BM_cu_conv_block_crack(benchmark::State& state) {
	pointi<2> ext;
	fill(ext, state.range(0));
	cu_tensor<_ValueType, 2> ts_src(ext);
	cu_tensor<_ValueType, 2> ts_re(ts_src.extent());
	fill(host_mask, 1.0f / host_mask.size());
	cuda::copy_symbol(host_mask, mask);

	while (state.KeepRunning()) {
		cuda::puzzle::conv_block_crack<16,16>(ts_src, ts_re);
	}

	auto bytes_size = static_cast<size_t>(ts_src.size()) * sizeof(_ValueType);
	state.SetBytesProcessed(state.iterations() * bytes_size * 2);
}

BENCHMARK_TEMPLATE1(BM_cu_conv_block_crack, float)->RangeMultiplier(2)->Range(128, 4096)->UseRealTime();




