﻿#include <matazure/tensor>

using namespace matazure;

int main(int argc, char *argv[]) {

	tensor<point<byte, 3>, 2> ts_rgb(512, 512);
	io::read_raw_data("data/lena_rgb888_512x512.raw_data", ts_rgb);
	
	auto t = ts_rgb[pointi<2>{2, 3}];

	return 0;
}
