﻿#include <matazure/tensor>
#include <matazure/cuda/puzzle/conv.hpp>

using namespace matazure;

__constant__ static_tensor<float,meta::array< 3,  3>> mask;
MATAZURE_PUZZEL_CONV_GLOBAL(conv_global, mask)
MATAZURE_PUZZEL_CONV_BLOCK(conv_block, mask)
MATAZURE_PUZZEL_CONV_BLOCK_WITH_CRACK(conv_block_crack, mask)

int main() {
	try {
		static_tensor<float,meta::array< 3,  3>> host_mask;
		fill(host_mask, 1.0f / host_mask.size());
		cuda::copy_symbol(host_mask, mask);

		tensor<byte, 2> gray(512, 512);
		io::read_raw_data("data/lena_gray8_512x512.raw_data", gray);
		auto cu_gray = mem_clone(gray, device_t{});

		auto lcts_conv = cuda::puzzle::conv_global(tensor_cast<float>(clamp_zero(cu_gray)));
		auto cts_conv = apply(lcts_conv, op::saturate_convert<byte>{}).persist();
		cuda::barrier();
		auto ts_conv = mem_clone(cts_conv, host_t{});
		io::write_raw_data("data/lena_gray8_conv_512x512.raw_data", ts_conv);

		auto cts_conv_block = cuda::puzzle::conv_block<16, 16>(tensor_cast<float>(cu_gray));
		auto cts_byte_conv_block = apply(cts_conv_block, op::saturate_convert<byte>{}).persist();
		cuda::barrier();
		auto ts_byte_conv_block = mem_clone(cts_byte_conv_block, host_t{});
		io::write_raw_data("data/lena_gray8_conv_block_512x512.raw_data", ts_byte_conv_block);

		auto cts_conv_block_crack = cuda::puzzle::conv_block_crack<32, 32>(tensor_cast<float>(clamp_zero(cu_gray)));
		auto cts_byte_conv_block_crack = apply(cts_conv_block_crack, op::saturate_convert<byte>{}).persist();
		cuda::barrier();
		auto ts_byte_conv_block_crack = mem_clone(cts_byte_conv_block_crack, host_t{});
		io::write_raw_data("data/lena_gray8_conv_block_crack_512x512.raw_data", ts_byte_conv_block_crack);
	}
	catch (std::exception &e) {
		std::cout << "unexpected error: " << e.what() << std::endl;
	}
}
